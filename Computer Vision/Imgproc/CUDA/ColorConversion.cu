
#include <hip/hip_runtime.h>
﻿// #include <cuda.h>
// #include <device_launch_parameters.h>
// #include <cuda_runtime_api.h>
//
// #include <iostream>
//
// __global__ void grayscale(uint8_t* d_src, uint8_t* d_dst, int rows, int cols, int channels)
// {
//     size_t i = threadIdx.x + blockIdx.x * blockDim.x;
//     size_t j = threadIdx.y + blockIdx.y * blockDim.y;
//
//     if(i >= rows || j >= cols)
//         return;
//
//     if(channels == 3)
//     {
//         // if()
//         uchar3 img = {d_src[i*cols + j], d_src[i*cols + j + 1], d_src[i*cols + j + 2]};
//     }
// }