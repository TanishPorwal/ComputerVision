
#include <hip/hip_runtime.h>
﻿// #include <cuda.h>
// #include <device_launch_parameters.h>
// #include <cuda_runtime_api.h>
// #include <cudnn.h>
// #include <cudnn_ops_infer.h>
//
// #include <iostream>
//
// #define checkCUDNN(expression)                               \
//   {                                                          \
//     cudnnStatus_t status = (expression);                     \
//     if (status != CUDNN_STATUS_SUCCESS) {                    \
//       std::cerr << "Error on line " << __LINE__ << ": "      \
//                 << cudnnGetErrorString(status) << std::endl; \
//       std::exit(EXIT_FAILURE);                               \
//     }                                                        \
//   }
//
//
// namespace cv
// {
//     __global__ void edge_detect_cuda(uint8_t& d_data, int rows, int cols)
//     {
//       cudnnHandle_t cudnnHandle;
//       checkCUDNN(cudnnCreate(&cudnnHandle));
//       
//       cudnnTensorDescriptor_t input_descriptor;
//       checkCUDNN(cudnnCreateTensorDescriptor(&input_descriptor));
//       checkCUDNN(cudnnSetTensor4dDescriptor(input_descriptor, CUDNN_TENSOR_NHWC, CUDNN_DATA_FLOAT, 1, 3, rows, cols));
//
//       cudnnTensorDescriptor_t output_descriptor;
//       checkCUDNN(cudnnCreateTensorDescriptor(&output_descriptor));
//       checkCUDNN(cudnnSetTensor4dDescriptor(output_descriptor, CUDNN_TENSOR_NHWC, CUDNN_DATA_FLOAT, 1, 3, rows, cols));
//
//       cudnnFilterDescriptor_t kernel_descriptor;
//       checkCUDNN(cudnnCreateFilterDescriptor(&kernel_descriptor));
//       checkCUDNN(cudnnSetFilter4dDescriptor(kernel_descriptor, CUDNN_DATA_FLOAT, CUDNN_TENSOR_NCHW, 3, 3, 3, 3));
//
//       
//     }
// }
